// %%cu
#include<bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;

const int numParticles = 1000;
const float timeStep = 0.1f;

struct Particle {
    float x, y, vx, vy;
};

// CUDA kernel for updating particle positions
__global__ void updateParticlesCUDA(Particle* particles, int numParticles, float timeStep) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < numParticles) {
        // updating the particle pos. by using velocity  if id is less tha numParticles
        particles[id].x += particles[id].vx * timeStep;
        particles[id].y += particles[id].vy * timeStep;

        particles[id].vx += 0.1f * sinf(particles[id].y);
        particles[id].vy -= 0.1f * cosf(particles[id].x);
    }
}

void initializeParticles(Particle* particles, int numParticles) {
    for (int i = 0; i < numParticles; ++i) {
        particles[i].x = static_cast<float>(rand()) / RAND_MAX;
        particles[i].y = static_cast<float>(rand()) / RAND_MAX;
        particles[i].vx = 0.1f * (static_cast<float>(rand()) / RAND_MAX - 0.5f);
        particles[i].vy = 0.1f * (static_cast<float>(rand()) / RAND_MAX - 0.5f);
    }
}

void writeParticlePositions(const Particle* particles, int numParticles, const string& filename) {
    ofstream outputFile(filename);

    for (int i = 0; i < numParticles; ++i) {
        outputFile << particles[i].x << " " << particles[i].y << '\n';
    }

    outputFile.close();
}

int main() {
    Particle* particles = new Particle[numParticles];
    Particle* d_particles;

    // Allocate device memory for particles
    hipMalloc((void**)&d_particles, numParticles * sizeof(Particle));

    // Initialize particles
    initializeParticles(particles, numParticles);

    // Copy particle data from host to device
    hipMemcpy(d_particles, particles, numParticles * sizeof(Particle), hipMemcpyHostToDevice);

    // Configure CUDA kernel launch parameters
    int threadsPerBlock = 256;
    int blocksPerGrid = (numParticles + threadsPerBlock - 1) / threadsPerBlock;

    // Run the simulation on the GPU for a number of time steps
    for (int step = 0; step < 1000; ++step) {
        updateParticlesCUDA<<<blocksPerGrid, threadsPerBlock>>>(d_particles, numParticles, timeStep);
        hipDeviceSynchronize(); // Ensure all CUDA operations are completed
    }

    // Copy particle data from device to host
    hipMemcpy(particles, d_particles, numParticles * sizeof(Particle), hipMemcpyDeviceToHost);

    // Store particle positions in the output file
    writeParticlePositions(particles, numParticles, "particle_positions.txt");

    // Free device memory
    hipFree(d_particles);

    delete[] particles;

    return 0;
}
